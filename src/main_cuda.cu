#include "hip/hip_runtime.h"
// src/main_cuda.cu

#include <stdio.h>
#include "solver_cuda.h"
#include <hip/hip_runtime.h>
#include "params.h"

#include "params.h"
#ifndef BLOCK_X
#define BLOCK_X 32        // defaults p/ build sem -DBLOCK_X
#endif
#ifndef BLOCK_Y
#define BLOCK_Y 32
#endif

#define CUDA_CHECK(msg)                                           \
    do {                                                          \
        hipError_t e = hipGetLastError();                       \
        if (e != hipSuccess) {                                   \
            fprintf(stderr, "CUDA-ERROR (%s): %s\n", msg,         \
                    hipGetErrorString(e));                       \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    } while (0)

// protótipos de kernels definidos em solver_cuda.cu
__global__ void initialize_kernel(double*, double*, double*);
__global__ void update_velocities_kernel(double*, double*, double*, double*, double*);
__global__ void solve_pressure_kernel(const double*, const double*, const double*, double*);

int main() {
    double *d_u, *d_v, *d_p, *d_p_new, *d_u_new, *d_v_new;
    size_t size = NX * NY * sizeof(double);

    hipMalloc(&d_u, size);  hipMalloc(&d_v, size);
    hipMalloc(&d_p, size);  hipMalloc(&d_p_new, size);
    hipMalloc(&d_u_new, size);  hipMalloc(&d_v_new, size);

    dim3 block(BLOCK_X, BLOCK_Y);
    dim3 grid((NX + BLOCK_X - 1) / BLOCK_X,
              (NY + BLOCK_Y - 1) / BLOCK_Y);
    initialize_kernel<<<grid, block>>>(d_u, d_v, d_p);
    CUDA_CHECK("init");
    hipDeviceSynchronize();

    hipEvent_t t0, t1;
    hipEventCreate(&t0);  hipEventCreate(&t1);
    hipEventRecord(t0);


    for (int t = 0; t < NT; ++t) {
        solve_pressure_kernel<<<grid, block>>>(d_u, d_v, d_p, d_p_new);
        CUDA_CHECK("pressure");

        update_velocities_kernel<<<grid, block>>>(d_u, d_v, d_p,
                                                d_u_new, d_v_new);
        CUDA_CHECK("vel");

        hipMemcpy(d_u, d_u_new, size, hipMemcpyDeviceToDevice);
        CUDA_CHECK("copy u");

        hipMemcpy(d_v, d_v_new, size, hipMemcpyDeviceToDevice);
        CUDA_CHECK("copy v");
    }

    hipEventRecord(t1);  hipEventSynchronize(t1);
    float ms; hipEventElapsedTime(&ms, t0, t1);
    printf("cuda:   %.9f\n", ms / 1e3);
    hipFree(d_u); hipFree(d_v); hipFree(d_p); hipFree(d_p_new);
    hipFree(d_u_new); hipFree(d_v_new);
    return 0;
}
