// src/main_cuda.cu

#include <stdio.h>
#include "solver_cuda.h"
#include <hip/hip_runtime.h>
#include "params.h"
#include <stdlib.h>
#include "helpers.h"

#ifndef BLOCK_X
#define BLOCK_X 32        // defaults p/ build sem -DBLOCK_X
#endif
#ifndef BLOCK_Y
#define BLOCK_Y 32
#endif

#include <stdio.h>
#include <stdlib.h>

#define CUDA_CHECK(msg)                                             \
    do {                                                            \
        hipError_t e = hipGetLastError();                         \
        if (e != hipSuccess) {                                     \
            fprintf(stderr, "CUDA-ERROR (%s): %s\n", (msg),         \
                    hipGetErrorString(e));                         \
            exit(EXIT_FAILURE);                                     \
        }                                                           \
    } while (0)

int main() {
    double *d_u, *d_v, *d_p, *d_p_new, *d_u_new, *d_v_new;
    size_t size = NX * NY * sizeof(double);

    // Pega envs
    const char *variant = getenv_or("WAVE_VARIANT", "cuda");
    const char *cfg     = getenv_or("WAVE_CFG",     "N100_T32_TB1");
    const char *param   = getenv_or("WAVE_PARAM",   "-");

    // Host buffer para salvar p
    double *p_host = (double*)malloc(size);

    hipMalloc(&d_u, size);  hipMalloc(&d_v, size);
    hipMalloc(&d_p, size);  hipMalloc(&d_p_new, size);
    hipMalloc(&d_u_new, size);  hipMalloc(&d_v_new, size);

    dim3 block(BLOCK_X, BLOCK_Y);
    dim3 grid((NX + BLOCK_X - 1) / BLOCK_X,
              (NY + BLOCK_Y - 1) / BLOCK_Y);

    initialize_kernel<<<grid, block>>>(d_u, d_v, d_p);
    CUDA_CHECK("init");
    hipDeviceSynchronize();

    hipEvent_t t0, t1;
    hipEventCreate(&t0);  hipEventCreate(&t1);
    hipEventRecord(t0);

    for (int t = 0; t < NT; ++t) {
        solve_pressure_kernel<<<grid, block>>>(d_u, d_v, d_p, d_p_new);
        CUDA_CHECK("pressure");

        update_velocities_kernel<<<grid, block>>>(d_u, d_v, d_p,
                                                  d_u_new, d_v_new);
        CUDA_CHECK("vel");

        hipMemcpy(d_u, d_u_new, size, hipMemcpyDeviceToDevice);
        CUDA_CHECK("copy u");
        hipMemcpy(d_v, d_v_new, size, hipMemcpyDeviceToDevice);
        CUDA_CHECK("copy v");
        hipMemcpy(d_p, d_p_new, size, hipMemcpyDeviceToDevice);
        CUDA_CHECK("copy p");

        if (t % 10 == 0) {
            hipMemcpy(p_host, d_p, size, hipMemcpyDeviceToHost);
            save_results_cuda(variant, cfg, param, t / 10, p_host);
        }
    }

    hipEventRecord(t1);  hipEventSynchronize(t1);
    float ms;
    hipEventElapsedTime(&ms, t0, t1);
    printf("cuda:   %.9f\n", ms / 1e3);

    // cleanup
    hipFree(d_u); hipFree(d_v); hipFree(d_p); hipFree(d_p_new);
    hipFree(d_u_new); hipFree(d_v_new);
    free(p_host);
    return 0;
}
